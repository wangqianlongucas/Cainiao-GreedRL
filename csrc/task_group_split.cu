#include "hip/hip_runtime.h"
#include "task_group_split.h"

__global__ void task_group_split_kernel(
        int* group, bool* value, bool* output,
        const int batch_size, const int task_num, const int group_num)
{
    group += blockIdx.x * task_num;
    value += blockIdx.x * task_num;
    extern __shared__ bool temp[];
    
    __shared__ bool split;
    if(threadIdx.x == 0) split = false;

    for(int i=threadIdx.x; i<group_num; i+=blockDim.x)
    {
        temp[i] = false;
    }

    __syncthreads();

    for(int i=threadIdx.x; i<task_num; i+=blockDim.x)
    {
        int g = group[i];
        if(value[i]) temp[g] = true; 
    }
    
    __syncthreads();
    
    for(int i=threadIdx.x; i<task_num; i+=blockDim.x)
    {
        int g = group[i];
        if(temp[g] && !value[i]) split = true;
    }
    
    __syncthreads();

    if(threadIdx.x == 0) output[blockIdx.x] = split;
};

void task_group_split_cuda(
        int* group, bool* value, bool* output,
        const int batch_size, const int task_num, const int group_num, const int device)
{
    const int shared_mem = group_num * sizeof(bool);

    GRL_CHECK_CUDA(hipSetDevice(device));

    task_group_split_kernel<<<batch_size, 256, shared_mem>>>(
        group, value, output, batch_size, task_num, group_num);

    GRL_CHECK_CUDA(hipGetLastError());
};

